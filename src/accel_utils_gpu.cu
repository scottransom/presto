#include "hip/hip_runtime.h"
//GPU functions for accelsearch
//by Jintao Luo, NRAO

// includes, CUDA
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include "hip/device_functions.h"

#include "accel_utils_gpu.h"

typedef float2 Complex;

//define a texture memory
texture<Complex> tex_d_kernel;
texture<Complex> tex_d_data;

texture<Complex> tex_d_result;

texture<Complex, 2> tex_d_result_2D;
hipChannelFormatDesc channelDesc ;

texture<unsigned short> tex_d_zinds;
texture<unsigned short> tex_d_rinds;

texture<float> tex_d_fundamental;

static __device__ __host__ inline fcomplex ComplexScale(fcomplex, float);
static __device__ __host__ inline fcomplex ComplexMul(fcomplex, fcomplex);
static __device__ __host__ inline fcomplex ComplexMul_02(fcomplex, fcomplex);

static __global__ void ComplexPointwiseMulAndScale_one_loop(fcomplex *c, fcomplex *a, const fcomplex *b, int numkern_in_array, int data_size, float scale, int kernel_array_offset);

static __global__ void ComplexPointwiseMulAndScale_one_loop_02(fcomplex *c, fcomplex *a, const fcomplex *b, int numkern_in_array, int data_size, float scale, int kernel_array_offset);

static __global__ void Complex_Pow_and_Chop(float *d_pow, fcomplex *d_result, int fftlen, int numkern_in_array, int chopbins, int numtocopy);

static __global__ void add_ffdotpows_on_gpu(float *d_fundamental, fcomplex *d_result, int numzs_full, int numrs_full, unsigned short *zinds, unsigned short *rinds, int fftlen, int chopbins, double obs_zlo, double fullrlo, double harm_fract, int zlo, int rlo);


extern "C"
extern int  search_ffdotpows_gpu(float powcut, float *d_fundamental, accel_cand_gpu * cand_array_search_gpu, accel_cand_gpu * cand_array_sort_gpu, int numzs, int numrs, accel_cand_gpu *cand_gpu_cpu);

static __global__ void  search_ffdotpows_kernel(float powcut, float *d_fundamental, accel_cand_gpu * cand_array_search_gpu, int numzs, int numrs, int *d_addr_jluo);

extern "C"
fcomplex * prep_data_on_gpu(subharminfo **subharminfs, int numharmstages);

extern "C"
fcomplex * prep_result_on_gpu(subharminfo **subharminfs, int numharmstages);

extern "C"
fcomplex * cp_kernel_array_to_gpu(subharminfo **subharminfs, int numharmstages, int **offset_array);

extern "C"
fcomplex * cp_input_to_gpu(fcomplex *input_vect_on_cpu, long long numbins, long long N);

extern "C"
unsigned short *prep_rz_inds_on_gpu(int size_inds);

extern "C"
float * prep_float_vect_on_gpu( int size);

extern "C"
accel_cand_gpu *prep_cand_array(int size);

extern "C"
void cudaFree_kernel_vect(fcomplex *in);

extern "C"
void select_cuda_dev(int cuda_inds);

extern "C"
void complex_corr_conv_gpu(fcomplex * data, fcomplex * kernel_vect_on_gpu,
                       int numdata, 
                       int numkern_in_array, 
                       int stage, int harmtosum, int harmnum, 
                       int ** offset_array, fcomplex *d_data, fcomplex *d_result,
                       int chopbins, int numtocopy,
                       unsigned short *zinds, unsigned short *rinds,
                       int numzs_full, int numrs_full,
                       float *d_fundamental,
                       unsigned short *d_zinds, unsigned short *d_rinds,
                       int datainf_flag,
                       presto_ffts ffts, presto_optype type,
                       double obs_zlo, double fullrlo, double harm_fract, int zlo, int rlo);


extern "C"
void init_cuFFT_plans(subharminfo **subharminfs, int numharmstages);

extern "C"
void destroy_cuFFT_plans(subharminfo **subharminfs, int numharmstages);

hipfftHandle plan_data_array[16][16];		
hipfftHandle plan_result_array[16][16];		

/******************************************** complex_corr_conv ********************************************************************/
void complex_corr_conv_gpu(fcomplex * data, fcomplex * kernel_vect_on_gpu,
                       int numdata, 
                       int numkern_in_array, 
                       int stage, int harmtosum, int harmnum, 
                       int ** offset_array, fcomplex *d_data, fcomplex *d_result,
                       int chopbins, int numtocopy,
                       unsigned short *zinds, unsigned short *rinds,
                       int numzs_full, int numrs_full,
                       float *d_fundamental,
                       unsigned short *d_zinds, unsigned short *d_rinds,
                       int datainf_flag,
                       presto_ffts ffts, presto_optype type,
                       double obs_zlo, double fullrlo, double harm_fract, int zlo, int rlo)
{


	int fftlen = numdata;
	int kernel_array_offset;

   if (ffts > 3) {
      printf("\nIllegal 'ffts' option (%d) in complex_corr_conv().\n", ffts);
      printf("Exiting.\n\n");
      exit(1);
   }
   if (type > 3) {
      printf("\nIllegal 'type' option (%d) in complex_corr_conv().\n", type);
      printf("Exiting.\n\n");
      exit(1);
   }
	
	if(harmtosum==1 && harmnum==1){
		kernel_array_offset = offset_array[0][0];
	}
	if(harmtosum > 1){
    kernel_array_offset = offset_array[stage][harmnum-1];
	}
	 
	//copy data to GPU memory
	checkCudaErrors(hipMemcpy(d_data, data, sizeof(fcomplex) * fftlen, hipMemcpyHostToDevice));
	//FFT data on GPU
	if(datainf_flag == 1)
	{					
		if(harmtosum==1 && harmnum==1){
			checkCudaErrors(hipfftExecC2C(plan_data_array[0][0], (hipfftComplex *)d_data, (hipfftComplex *)d_data, HIPFFT_FORWARD));
		}
		if(harmtosum > 1){
  	  checkCudaErrors(hipfftExecC2C(plan_data_array[stage][harmnum-1], (hipfftComplex *)d_data, (hipfftComplex *)d_data, HIPFFT_FORWARD));
		}

	}

	//Bind data and kernel to Texture Memory
	hipBindTexture(NULL, tex_d_kernel, kernel_vect_on_gpu, sizeof(fcomplex) * ( kernel_array_offset + 1 + numkern_in_array * fftlen ) );
	hipBindTexture(NULL, tex_d_data, d_data, sizeof(fcomplex) * fftlen);
	//Mul the FFTed data with Kernels
	if (type == CORR || type == INPLACE_CORR) {
		ComplexPointwiseMulAndScale_one_loop<<<512, 512>>>(d_result, d_data, kernel_vect_on_gpu, numkern_in_array, fftlen, 1.0/fftlen, kernel_array_offset);
	}
	else {
		ComplexPointwiseMulAndScale_one_loop_02<<<512, 512>>>(d_result, d_data, kernel_vect_on_gpu, numkern_in_array, fftlen, 1.0/fftlen, kernel_array_offset);
	}	
	//unbind the data and kenerl from Texture memory
	hipUnbindTexture(tex_d_kernel);
	hipUnbindTexture(tex_d_data);  	

   
  //Inverse FFT   
	if(harmtosum==1 && harmnum==1){
		checkCudaErrors(hipfftExecC2C(plan_result_array[0][0], (hipfftComplex *)d_result, (hipfftComplex *)d_result, HIPFFT_BACKWARD));	
	}
	if(harmtosum > 1){
 	  checkCudaErrors(hipfftExecC2C(plan_result_array[stage][harmnum-1], (hipfftComplex *)d_result, (hipfftComplex *)d_result, HIPFFT_BACKWARD));	
	} 

	//bind the FFTed result
		//hipBindTexture(NULL, tex_d_result, d_result, sizeof(fcomplex) * fftlen * numkern_in_array );	
		//bind d_result to its 2D texture memory
		checkCudaErrors(hipBindTexture2D(NULL, tex_d_result_2D, d_result, channelDesc, fftlen, numkern_in_array, sizeof(fcomplex) * fftlen ));
	//sum harmonics
	if(harmtosum==1 && harmnum==1){//if fundamental
		Complex_Pow_and_Chop<<<512, 512>>>(d_fundamental, d_result, fftlen, numkern_in_array, chopbins, numtocopy);
	}
	if(harmtosum > 1){	//if harmonics
		//move zinds and rinds to GPU
		checkCudaErrors(hipMemcpy(d_zinds, zinds, sizeof(unsigned short) * numzs_full, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_rinds, rinds, sizeof(unsigned short) * numrs_full, hipMemcpyHostToDevice));		
		//bind zinds and rinds to Texture Memory
		hipBindTexture(NULL, tex_d_zinds, d_zinds, sizeof(unsigned short) * numzs_full );
		hipBindTexture(NULL, tex_d_rinds, d_rinds, sizeof(unsigned short) * numrs_full );		
		//add_ffdotpows_on_gpu
		add_ffdotpows_on_gpu<<<512, 512>>>(d_fundamental, d_result, numzs_full, numrs_full, d_zinds, d_rinds, fftlen, chopbins, obs_zlo, fullrlo, harm_fract, zlo, rlo);
		//Unbind zinds and rinds from Texture Memory		
		hipUnbindTexture(tex_d_zinds);    
		hipUnbindTexture(tex_d_rinds);    
	}
 
	//hipUnbindTexture(tex_d_result);
	checkCudaErrors(hipUnbindTexture(tex_d_result_2D));
  
}                   

/******************************************** add fftdot pows on GPU, choping included ************************************************/
static __global__ void add_ffdotpows_on_gpu(float *d_fundamental, fcomplex *d_result, int numzs_full, int numrs_full, unsigned short *zinds, unsigned short *rinds, int fftlen, int chopbins, double obs_zlo, double fullrlo, double harm_fract, int zlo, int rlo)
{

    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;		
    
    //int addr_z, addr_r, addr_result, addr_fundamental ;
    int addr_z, addr_r ;				
    int addr_result ;
		
		Complex buf;
		
		int z_index, r_index;
		
		for (int i = threadID; i < numrs_full * numzs_full; i += numThreads){    		
						
				z_index = i/numrs_full ;
				addr_z = tex1Dfetch(tex_d_zinds, z_index) ;	
				
				r_index = i -  z_index * numrs_full ;				
				addr_r = tex1Dfetch(tex_d_rinds, r_index) ;
				
				//addr_result = addr_z * fftlen + chopbins + addr_r ;
				//buf = tex1Dfetch(tex_d_result, addr_result) ;    		
				
				buf = tex2D(tex_d_result_2D, chopbins + addr_r, addr_z);
				
				d_fundamental[i] += buf.x * buf.x + buf.y * buf.y ;
				
		}

}




/********************************************* Calculate complex ffdot pows, choping included *****************************************/
static __global__ void Complex_Pow_and_Chop(float *d_pow, fcomplex *d_result, int fftlen, int numkern_in_array, int chopbins, int numtocopy)
/*
	calaulate POW of data in d_result
	d_pw:				where the result stored
	d_result:		the array contains data to Pow_and_chop
	fftlen:			length of FFT used in the complex_corr_conv_gpu
	numkern_in_array:	num of kernels in array in complex_corr_conv_gpu
	chopbins:		num of bins that should be discarded in the head and tail of the result vectors
	numtocopy:	num of data should be kept in each vector of d_result, after the first chopbins points	
*/
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;		
    
    int addr_result;
    
    Complex buf;
    
    int z_ind, r_ind ;
    
    for (int i = threadID; i < numkern_in_array*numtocopy; i += numThreads){
    	
    	z_ind = i/numtocopy;
    	r_ind = i - z_ind * numtocopy;
    	
    	//addr_result = z_ind * fftlen + chopbins +  r_ind;
    	
    	//buf = tex1Dfetch(tex_d_result, addr_result) ;
    
    	buf = tex2D(tex_d_result_2D, chopbins + r_ind, z_ind);
    
    	d_pow[i] = buf.x * buf.x + buf.y * buf.y ;
    }   

}


/**********************************************************************************************************************************************************/
//complex operations

// Complex scale
static __device__ __host__ inline fcomplex ComplexScale(fcomplex a, float s)
{
    fcomplex c;
    c.r = s * a.r;
    c.i = s * a.i;
    return c;
}

// Complex multiplication
static __device__ __host__ inline fcomplex ComplexMul(fcomplex a, fcomplex b)
{
    fcomplex c;
    c.r = a.r * b.r + a.i * b.i;
    c.i = a.i * b.r - a.r * b.i;
    return c;
}

static __device__ __host__ inline fcomplex ComplexMul_02(fcomplex a, fcomplex b)
{
    fcomplex c;
    c.r = a.r * b.r - a.i * b.i;
    c.i = a.i * b.r + a.r * b.i;
    return c;
}


//Use one loop to realize Complex Point Mul and Scale 
static __global__ void ComplexPointwiseMulAndScale_one_loop(fcomplex *c, fcomplex *a, const fcomplex *b, int numkern_in_array, int data_size, float scale, int kernel_array_offset)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;		
		
		fcomplex a_buf ;
		fcomplex b_buf ;
		Complex buf_buf_a, buf_buf_b;
		int i;
		
		int a_index, b_index;
		
		int total_num = data_size*numkern_in_array;
		
		for (i = threadID; i < total_num; i += numThreads)
		{
				a_index = i/data_size;
				a_index = i - a_index * data_size ;
				buf_buf_a = tex1Dfetch(tex_d_data, a_index);
    		a_buf.r = buf_buf_a.x;
    		a_buf.i = buf_buf_a.y;
    		
    		b_index = i + kernel_array_offset;
    		buf_buf_b = tex1Dfetch(tex_d_kernel, b_index) ;
				
				b_buf.r = buf_buf_b.x;
	    	b_buf.i = buf_buf_b.y;
        c[i] = ComplexScale(ComplexMul(a_buf, b_buf), scale);                                
        	
        
		}

}

static __global__ void ComplexPointwiseMulAndScale_one_loop_02(fcomplex *c, fcomplex *a, const fcomplex *b, int numkern_in_array, int data_size, float scale, int kernel_array_offset)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;		
		
		fcomplex a_buf ;
		fcomplex b_buf ;
		Complex buf_buf_a, buf_buf_b;
		int i;

		int a_index, b_index;
		
		int total_num = data_size*numkern_in_array;
		
		for (i = threadID; i < total_num; i += numThreads)
		{
				a_index = i/data_size;
				a_index = i - a_index * data_size ;
				buf_buf_a = tex1Dfetch(tex_d_data, a_index);
    		a_buf.r = buf_buf_a.x;
    		a_buf.i = buf_buf_a.y;
    		
    		b_index = i + kernel_array_offset;
    		buf_buf_b = tex1Dfetch(tex_d_kernel, b_index) ;
				
				b_buf.r = buf_buf_b.x;
	    	b_buf.i = buf_buf_b.y;
        c[i] = ComplexScale(ComplexMul_02(a_buf, b_buf), scale);                                
        	
        
		}		

}
/**********************************************************************************************************************************************************/

//-------------------------Prepare search_ffdotpows cand array
accel_cand_gpu *prep_cand_array(int size)
{
	
	accel_cand_gpu * cand_on_gpu;
	checkCudaErrors(hipMalloc((void **)&cand_on_gpu, size * sizeof(accel_cand_gpu)));
	
	return cand_on_gpu ;
	
}

int  search_ffdotpows_gpu(float powcut, float *d_fundamental, accel_cand_gpu * cand_array_search_gpu, accel_cand_gpu * cand_array_sort_gpu, int numzs, int numrs, accel_cand_gpu *cand_gpu_cpu)
{
	
	int *d_addr;
	int h_addr;
	
	checkCudaErrors(hipMalloc((void **)&d_addr, sizeof(int) * 1));	
	checkCudaErrors(hipMemset(d_addr, 0, sizeof(int))); // set d_addr to 0

	//bind d_fundamental to texture
	hipBindTexture(NULL, tex_d_fundamental, d_fundamental, sizeof(float) * numzs * numrs );

	//search ffdot_pow
	search_ffdotpows_kernel<<<512, 512>>>(powcut, d_fundamental, cand_array_search_gpu, numzs, numrs, d_addr);

	//get nof_cand
	checkCudaErrors(hipMemcpy(&h_addr, d_addr, sizeof(int) * 1, hipMemcpyDeviceToHost));	
	int nof_cand ;		
	nof_cand = h_addr;		

	//get the candicates
	checkCudaErrors(hipMemcpy(cand_gpu_cpu, cand_array_search_gpu, sizeof(accel_cand_gpu) * nof_cand, hipMemcpyDeviceToHost));
	
	hipFree(d_addr);
	
	hipUnbindTexture(tex_d_fundamental);  
	
	return nof_cand ;
}

static __global__ void  search_ffdotpows_kernel(float powcut, float *d_fundamental, accel_cand_gpu * cand_array_search_gpu, int numzs, int numrs, int *d_addr_jluo)
{

    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;		

		int i ;		
		int nof_cand = 0;

		float pow ;
				
		int addr_search=0;	

		accel_cand_gpu cand_tmp ;
		
		int z_ind, r_ind;

		for (i = threadID; i < numzs*numrs; i += numThreads)
    {    	
    
    	nof_cand = 0;   	    	

   		pow = tex1Dfetch(tex_d_fundamental, i);

    		if(pow > powcut)
    		{    			
    			cand_tmp.pow = pow ;
    			nof_cand += 1 ;
    			
    			cand_tmp.nof_cand = nof_cand ;
    			
    			z_ind = (int)(i/numrs);
    			cand_tmp.z_ind = z_ind;
    			
    			r_ind = i - z_ind * numrs ;
    			cand_tmp.r_ind = r_ind;  			

    			addr_search = atomicAdd(&d_addr_jluo[0], 1);    			
    			
    			cand_array_search_gpu[ addr_search ] = cand_tmp ;    			   			    			
    		}
    }
}

//-------------------------Prepare vectors for zinds and rinds
unsigned short *prep_rz_inds_on_gpu(int size_inds)
{

	unsigned short *inds;
	checkCudaErrors(hipMalloc((void **)&inds, size_inds * sizeof(unsigned short)));

	return inds;

}

//-------------------------Prepare a float-point vectors 
float * prep_float_vect_on_gpu( int size)
{

	float * f_vect;
	checkCudaErrors(hipMalloc((void **)&f_vect, size * sizeof(float)));
	
	return f_vect;

}

//----------Copy input data to GPU---------------------------------
fcomplex * cp_input_to_gpu(fcomplex *input_vect_on_cpu, long long numbins, long long N)
{
	fcomplex *input_vect_on_gpu;		
	
  checkCudaErrors(hipMalloc((void **)&input_vect_on_gpu, sizeof(fcomplex) * numbins));       		
	
	//check some parameters
	printf("\nNum of data points: %d\n", N);
	printf("\nNum of bins : %d\n", numbins);	
	
	checkCudaErrors(hipMemcpy( input_vect_on_gpu, input_vect_on_cpu, sizeof(fcomplex) * numbins, hipMemcpyHostToDevice) );
	
	return input_vect_on_gpu;
}

//----------Prepare d_data on GPU---------------------
fcomplex * prep_data_on_gpu(subharminfo **subharminfs, int numharmstages)
//hipMalloc d_data for the whole search process
{

	int harm, harmtosum, stage;		
	int fftlen;
	int size_d_data;
	
	fcomplex *d_data;

	size_d_data = 0;
		
	fftlen = subharminfs[0][0].kern[0].fftlen;
	
	size_d_data = fftlen ;

if (numharmstages > 1) {

	for(stage=1; stage<numharmstages; stage++){	
		harmtosum = 1 << stage;		
		for (harm = 1; harm < harmtosum; harm += 2) {               					
			
			fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
			
			if( size_d_data < fftlen )
			{
				size_d_data = fftlen;
			}
			
		}	
	}

}
	
	//alloc memory for device data
	 checkCudaErrors(hipMalloc((void **)&d_data, sizeof(fcomplex) * size_d_data));
	
	return d_data ;	
}                            

//----------Prepare d_result on GPU---------------------
fcomplex * prep_result_on_gpu(subharminfo **subharminfs, int numharmstages)
//hipMalloc d_result for the whole search process
{

	int harm, harmtosum, stage;		
	int numkern, fftlen;
	int size_d_result;
	
	fcomplex *d_result;	
	size_d_result = 0;

	numkern = subharminfs[0][0].numkern;
	fftlen = subharminfs[0][0].kern[0].fftlen;

	size_d_result = numkern * fftlen;

if (numharmstages > 1) {

	for(stage=1; stage<numharmstages; stage++){	
		harmtosum = 1 << stage;		
		for (harm = 1; harm < harmtosum; harm += 2) {               		
						
			numkern = subharminfs[stage][harm-1].numkern;
			fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
								
			if(size_d_result < numkern * fftlen )
			{
				size_d_result = numkern * fftlen;
			}			
		}	
	}	
}
	//Alloc mem for result on GPU  
  checkCudaErrors(hipMalloc((void **)&d_result, size_d_result * sizeof(fcomplex)));  
	
	//also prepare its 2D texture memory
	channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
	tex_d_result_2D.addressMode[0] = hipAddressModeClamp;
  tex_d_result_2D.addressMode[1] = hipAddressModeClamp;
  //texRef2D.filterMode = hipFilterModePoint;
  tex_d_result_2D.filterMode = hipFilterModePoint;
  tex_d_result_2D.normalized = false;
	
	return d_result ;
	
}                            

//----------Copy kernek array to GPU-------------------------------
fcomplex * cp_kernel_array_to_gpu(subharminfo **subharminfs, int numharmstages, int **offset_array)
//cp kernel arrays contained in **subharminfs to GPU memory *kernel_array_on_gpu
//and store the offset within *kernel_array_on_gpu for every array, offsets stored in **offset_array
//subharminfs: input
//numharmstages: input
//offset_array: output
//kernel_array_on_gpu: output
//
{

	int harm, harmtosum, stage;		
	int ii, jj;
	int kernel_total_size =0 ;
	int numkern, fftlen;
	unsigned int offset_base = 0, offset_tmp;
	fcomplex *kernel_vect_host;
	fcomplex *kernel_vect_on_gpu;
	
	//printf("cp_kernel_array_to_gpu: Check CUDA devices\n");
	
	numkern = subharminfs[0][0].numkern;
	fftlen = subharminfs[0][0].kern[0].fftlen;
	offset_array[0][0] = kernel_total_size ; 
	kernel_total_size = kernel_total_size + numkern * fftlen;
	
	for(stage=1; stage<numharmstages; stage++){	
		harmtosum = 1 << stage;		
		for (harm = 1; harm < harmtosum; harm += 2) {               					
			numkern = subharminfs[stage][harm-1].numkern;
			fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
			offset_array[stage][harm-1] = kernel_total_size ; 
			kernel_total_size = kernel_total_size + numkern * fftlen;
		}	
	}	
	//printf("    ----Total size of kernel arrays is : %d\n", kernel_total_size);
	checkCudaErrors(hipMalloc((void **)&kernel_vect_on_gpu, sizeof(fcomplex) * kernel_total_size));       
	
	kernel_vect_host = (fcomplex *)malloc( sizeof(fcomplex) * kernel_total_size);
	//take kernel_vect_host as buffer
	 //stage : 0
	 numkern = subharminfs[0][0].numkern;
   fftlen = subharminfs[0][0].kern[0].fftlen;
   offset_base = offset_array[0][0];
   for(ii=0; ii<numkern; ii++)
   	{   		
   		for(jj=0; jj<fftlen; jj++){   		
   			offset_tmp = offset_base + fftlen*ii + jj;
   			kernel_vect_host[offset_tmp].r = subharminfs[0][0].kern[ii].data[jj].r;
   			kernel_vect_host[offset_tmp].i = subharminfs[0][0].kern[ii].data[jj].i;
   		}
   	}

   //other stages
if (numharmstages > 1) {   
   for(stage=1; stage<numharmstages; stage++){	
			harmtosum = 1 << stage;		
			for(harm = 1; harm < harmtosum; harm += 2) {         

				offset_base = offset_array[stage][harm-1];

				fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
				numkern = subharminfs[stage][harm-1].numkern;

				for(ii=0; ii<numkern; ii++){
					for(jj=0; jj<fftlen; jj++){
						offset_tmp = offset_base + fftlen*ii + jj ;
						kernel_vect_host[offset_tmp].r = subharminfs[stage][harm-1].kern[ii].data[jj].r;
		   			kernel_vect_host[offset_tmp].i = subharminfs[stage][harm-1].kern[ii].data[jj].i;
						}
					}
					
				}
			}
}   
   checkCudaErrors(hipMemcpy( kernel_vect_on_gpu, kernel_vect_host, sizeof(fcomplex) * kernel_total_size, hipMemcpyHostToDevice) );

   free(kernel_vect_host);
   
   return kernel_vect_on_gpu;
	
}

//----------------------initialize cuFFT plans ----------------------
void init_cuFFT_plans(subharminfo **subharminfs, int numharmstages)
{
	int harm, harmtosum, stage;		
	int numkern, fftlen;
	
	printf("\ninit_cuFFT_plans\n");
	
	numkern = subharminfs[0][0].numkern;
	fftlen = subharminfs[0][0].kern[0].fftlen;

	checkCudaErrors(hipfftPlan1d(&plan_data_array[0][0], fftlen, HIPFFT_C2C, 1));		
	checkCudaErrors(hipfftPlan1d(&plan_result_array[0][0], fftlen, HIPFFT_C2C, numkern));		

	printf("stage: 0, fftlen : %d, numkern: %d\n", fftlen, numkern);

if (numharmstages > 1) {

	for(stage=1; stage<numharmstages; stage++){	
		harmtosum = 1 << stage;		
		for (harm = 1; harm < harmtosum; harm += 2) {               					
			numkern = subharminfs[stage][harm-1].numkern;
			fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
			
			printf("stage: %d, fftlen : %d, numkern: %d\n", stage, fftlen, numkern);
				
			checkCudaErrors(hipfftPlan1d(&plan_data_array[stage][harm-1], fftlen, HIPFFT_C2C, 1));		
			checkCudaErrors(hipfftPlan1d(&plan_result_array[stage][harm-1], fftlen, HIPFFT_C2C, numkern));		
						
		}	
	}	

}
	
}

//----------------------destroy cuFFT plans ----------------------
void destroy_cuFFT_plans(subharminfo **subharminfs, int numharmstages)
{
	int harm, harmtosum, stage;		
	int numkern, fftlen;
	
	printf("\ndestroy_cuFFT_plans\n");
	
	numkern = subharminfs[0][0].numkern;
	fftlen = subharminfs[0][0].kern[0].fftlen;

	checkCudaErrors(hipfftDestroy(plan_data_array[0][0]));
	checkCudaErrors(hipfftDestroy(plan_result_array[0][0]));

	printf("stage: 0, fftlen : %d, numkern: %d\n", fftlen, numkern);

if (numharmstages > 1) {

	for(stage=1; stage<numharmstages; stage++){	
		harmtosum = 1 << stage;		
		for (harm = 1; harm < harmtosum; harm += 2) {               					
			numkern = subharminfs[stage][harm-1].numkern;
			fftlen = subharminfs[stage][harm-1].kern[0].fftlen;
			
			printf("stage: %d, fftlen : %d, numkern: %d\n", stage, fftlen, numkern);
	
			checkCudaErrors(hipfftDestroy(plan_data_array[stage][harm-1]));
			checkCudaErrors(hipfftDestroy(plan_result_array[stage][harm-1]));
					
		}	
	}	
}
	
}


//----------------------select a cpu to play with --------------------
void select_cuda_dev(int cuda_inds)
{
	
	hipSetDevice(cuda_inds);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cuda_inds);

  printf("\nGPU Device %d: \"%s\" with Capability: %d.%d\n", cuda_inds, deviceProp.name, deviceProp.major, deviceProp.minor);
	
	hipDeviceReset();
}

//-------------------------clean up on GPU ----------------------------
void cudaFree_kernel_vect(fcomplex *in)
{
	hipFree( in );
}
